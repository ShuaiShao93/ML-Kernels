#include "hip/hip_runtime.h"
#include "gemm.h"

#include <cstdio>
#include <iostream>

__global__ void gemm_kernel_0(const float *a_ptr, const float *b_ptr,
                              float *c_ptr, int M, int N, int K, int stride_am,
                              int stride_ak, int stride_bk, int stride_bn,
                              int stride_cm, int stride_cn) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= M || y >= N) {
    return;
  }

  float accum = 0;
  for (int k = 0; k < K; k++) {
    accum += a_ptr[x * stride_am + k * stride_ak] *
             b_ptr[k * stride_bk + y * stride_bn];
  }
  c_ptr[x * stride_cm + y * stride_cn] = accum;
}

__global__ void gemm_kernel_1(const float *a_ptr, const float *b_ptr,
                              float *c_ptr, int M, int N, int K, int stride_am,
                              int stride_ak, int stride_bk, int stride_bn,
                              int stride_cm, int stride_cn) {
  // threadIdx.x is consecutive and should read contiguous memory.
  int m = blockIdx.y * blockDim.y + threadIdx.y;
  int n = blockIdx.x * blockDim.x + threadIdx.x;

  if (m >= M || n >= N) {
    return;
  }

  float accum = 0;
  for (int k = 0; k < K; k++) {
    accum += a_ptr[m * stride_am + k * stride_ak] *
             b_ptr[k * stride_bk + n * stride_bn];
  }
  c_ptr[m * stride_cm + n * stride_cn] = accum;
}

// Unfortunately blockDim is not constexpr.
template <const int BLOCK_SIZE_M, const int BLOCK_SIZE_N,
          const int BLOCK_SIZE_K>
__global__ void gemm_kernel_2(const float *a_ptr, const float *b_ptr,
                              float *c_ptr, int M, int N, int K, int stride_am,
                              int stride_ak, int stride_bk, int stride_bn,
                              int stride_cm, int stride_cn) {
  // Block offsets.
  int block_m = blockIdx.y * blockDim.y, block_n = blockIdx.x * blockDim.x;
  int thread_m = threadIdx.y, thread_n = threadIdx.x;
  a_ptr += block_m * stride_am;
  b_ptr += block_n * stride_bn;
  c_ptr += block_m * stride_cm + block_n * stride_cn;

  __shared__ float as[BLOCK_SIZE_M][BLOCK_SIZE_K],
      bs[BLOCK_SIZE_K][BLOCK_SIZE_N];
  float accu = 0;

  for (int block_k = 0; block_k < K; block_k += BLOCK_SIZE_K) {
    // Assumes BLOCK_SIZE_M and BLOCK_SIZE_N are not less than BLOCK_SIZE_K
    if (thread_n < BLOCK_SIZE_K) {
      as[thread_m][thread_n] =
          a_ptr[thread_m * stride_am + thread_n * stride_ak];
    }
    if (thread_m < BLOCK_SIZE_K) {
      bs[thread_m][thread_n] =
          b_ptr[thread_m * stride_bk + thread_n * stride_bn];
    }

    a_ptr += BLOCK_SIZE_K * stride_ak;
    b_ptr += BLOCK_SIZE_K * stride_bk;

    __syncthreads();

    // Very critical. In first branch, the loop can be unrolled.
    // We can also have an if condition in the loop, but it would be
    // worse because of the branch.
    if (block_k + BLOCK_SIZE_K < K) {
      for (int k = 0; k < BLOCK_SIZE_K; k++) {
        accu += as[thread_m][k] * bs[k][thread_n];
      }
    } else {
      for (int k = 0; k < K - block_k; k++) {
        accu += as[thread_m][k] * bs[k][thread_n];
      }
    }

    __syncthreads();
  }

  if (block_m + thread_m < M && block_n + thread_n < N) {
    c_ptr[thread_m * stride_cm + thread_n * stride_cn] = accu;
  }
}

template <int BM, int BN, int BK, int TM>
__global__ void gemm_kernel_3(const float *A, const float *B, float *C, int M,
                              int N, int K, int stride_am, int stride_ak,
                              int stride_bk, int stride_bn, int stride_cm,
                              int stride_cn) {
  int block_m = blockIdx.y * BM, block_n = blockIdx.x * BN;
  int thread_m = threadIdx.y, thread_n = threadIdx.x;
  A += block_m * stride_am;
  B += block_n * stride_bn;
  C += block_m * stride_cm + block_n * stride_cn;

  __shared__ float As[BM][BK], Bs[BK][BN];
  float accum[TM] = {0.};
  // Assumes thread block size M/N are not less than BK
  for (int block_k = 0; block_k < K; block_k += BK) {
    for (int m = thread_m * TM; m < thread_m * TM + TM; m += 1) {
      if (thread_n < BK) {
        As[m][thread_n] = A[m * stride_am + thread_n * stride_ak];
      }
    }
    if (thread_m < BK) {
      Bs[thread_m][thread_n] = B[thread_m * stride_bk + thread_n * stride_bn];
    }

    __syncthreads();

    A += BK * stride_ak;
    B += BK * stride_bk;

    // Assumes K is multiple of BK.
    for (int k = 0; k < BK; k += 1) {
      float b_tmp = Bs[k][thread_n];
      for (int m = thread_m * TM; m < thread_m * TM + TM; m++) {
        accum[m - thread_m * TM] += As[m][k] * b_tmp;
      }
    }

    __syncthreads();
  }

  for (int m = thread_m * TM; m < thread_m * TM + TM; m++) {
    if (block_m + m < M && block_n + thread_n < N) {
      C[m * stride_cm + thread_n * stride_cn] = accum[m - thread_m * TM];
    }
  }
}

void gemm(const float *a_ptr, const float *b_ptr, float *c_ptr, int M, int N,
          int K, int stride_am, int stride_ak, int stride_bk, int stride_bn,
          int stride_cm, int stride_cn, hipStream_t stream, int kernel_id) {
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, stream);

  switch (kernel_id) {
  case 0: {
    std::cout << "Using kernel " << kernel_id << ": naiive" << std::endl;
    dim3 gridDim((M + 32 - 1) / 32, (N + 32 - 1) / 32);
    dim3 blockDim(32, 32);
    gemm_kernel_0<<<gridDim, blockDim, 0, stream>>>(
        a_ptr, b_ptr, c_ptr, M, N, K, stride_am, stride_ak, stride_bk,
        stride_bn, stride_cm, stride_cn);
    break;
  }
  case 1: {
    std::cout << "Using kernel " << kernel_id << ": memory_colescing"
              << std::endl;
    dim3 gridDim((N + 32 - 1) / 32, (M + 32 - 1) / 32);
    dim3 blockDim(32, 32);
    gemm_kernel_1<<<gridDim, blockDim, 0, stream>>>(
        a_ptr, b_ptr, c_ptr, M, N, K, stride_am, stride_ak, stride_bk,
        stride_bn, stride_cm, stride_cn);
    break;
  }
  case 2: {
    constexpr int block_size = 32;
    std::cout << "Using kernel " << kernel_id << ": shared_memory" << std::endl;
    dim3 gridDim((N + block_size - 1) / block_size,
                 (M + block_size - 1) / block_size);
    dim3 blockDim(block_size, block_size);
    gemm_kernel_2<block_size, block_size, block_size>
        <<<gridDim, blockDim, 0, stream>>>(a_ptr, b_ptr, c_ptr, M, N, K,
                                           stride_am, stride_ak, stride_bk,
                                           stride_bn, stride_cm, stride_cn);
    break;
  }
  case 3: {
    constexpr int bm = 64, bn = 64, bk = 8, tm = 8;
    std::cout << "Using kernel " << kernel_id << ": Register 1D Cache" << std::endl;
    dim3 gridDim((N + bn - 1) / bn, (M + bm - 1) / bm);
    dim3 blockDim(bn, bm / tm);
    gemm_kernel_3<bm, bn, bk, tm><<<gridDim, blockDim>>>(
        a_ptr, b_ptr, c_ptr, M, N, K, stride_am, stride_ak, stride_bk,
        stride_bn, stride_cm, stride_cn);
    break;
  }

  default:
    std::cerr << "Invalid kernel id " << kernel_id;
  }

  hipEventRecord(end, stream);
  hipEventSynchronize(end);
  float time;
  hipEventElapsedTime(&time, start, end);
  std::cout << "Elaspsed time: " << time << std::endl;
}